#include "hip/hip_runtime.h"
#include "mex.h"
#include "matrix.h"
#include "gpu/mxGPUArray.h"
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <iostream>
#include "kernel_add.h"

#define MAX(a,b) (((a) > (b)) ? (a) : (b))
#define MIN(a,b) (((a) < (b)) ? (a) : (b))
#define ABS(x) ((x) > 0 ? (x) : -(x))
#define PI 3.141592653589793
// Set thread block size
#define BLOCKWIDTH 16
#define BLOCKHEIGHT 16 
#define BLOCKDEPTH 4

void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[]){
float *h_proj, *h_proj1;
int na, nb, iv;
h_proj1 = (float*)mxGetData(prhs[0]);
h_proj = (float*)mxGetData(prhs[1]);
iv = (int)mxGetScalar(prhs[2]);
na = (int)mxGetScalar(prhs[3]);
nb = (int)mxGetScalar(prhs[4]);

float *d_proj1, *d_proj;
hipMalloc((void**)&d_proj, na * nb * sizeof(float));
hipMalloc((void**)&d_proj1, na * nb * sizeof(float));
hipMemcpy(d_proj, h_proj, na * nb * sizeof(float), hipMemcpyHostToDevice);
hipMemcpy(d_proj1, h_proj1, na * nb * sizeof(float), hipMemcpyHostToDevice);
const dim3 gridSize((na + BLOCKWIDTH - 1) / BLOCKWIDTH, (nb + BLOCKHEIGHT - 1) / BLOCKHEIGHT, (1 + BLOCKDEPTH - 1) / BLOCKDEPTH);
const dim3 blockSize(BLOCKWIDTH,BLOCKHEIGHT, BLOCKDEPTH);
kernel_add<<<gridSize, blockSize>>>(d_proj1, d_proj, iv, na, nb, -1);
hipMemcpy(h_proj1, d_proj1, na * nb * sizeof(float), hipMemcpyDeviceToHost);
hipFree(d_proj);
hipFree(d_proj1);
//Reset device for profiling
hipDeviceReset();
return;
}


// #include "mex.h"
// #include "matrix.h"
// #include "gpu/mxGPUArray.h"
// #include <stdlib.h>
// #include <hip/hip_runtime.h>
// #include <math.h>
// #include <iostream>
// // #include "kernel_add.h"

// #define MAX(a,b) (((a) > (b)) ? (a) : (b))
// #define MIN(a,b) (((a) < (b)) ? (a) : (b))
// #define ABS(x) ((x) > 0 ? (x) : -(x))
// #define PI 3.141592653589793
// // Set thread block size
// #define BLOCKWIDTH 16
// #define BLOCKHEIGHT 16 
// #define BLOCKDEPTH 4

// __global__ void kernel_add(hipArray *proj1, hipArray *proj, int iv, int na, int nb, float weight);

// void mexFunction(int nlhs, mxArray *plhs[],
//                  int nrhs, mxArray const *prhs[]){
// float *h_proj, *h_proj1;
// int na, nb, iv;
// h_proj1 = (float*)mxGetData(prhs[0]);
// h_proj = (float*)mxGetData(prhs[1]);
// iv = (int)mxGetScalar(prhs[2]);
// na = (int)mxGetScalar(prhs[3]);
// nb = (int)mxGetScalar(prhs[4]);

// // pitched memory
// hipPitchedPtr p_proj, p_proj1;
// p_proj = make_hipPitchedPtr((void*)h_proj, na * sizeof(float), nb, 1);
// p_proj1 = make_hipPitchedPtr((void*)h_proj1, na * sizeof(float), nb, 1);

// hipArray *d_proj1, *d_proj;
// hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
// struct hipExtent extent = make_hipExtent(na, nb, 1);
// hipMalloc3DArray(&d_proj, &channelDesc, extent);
// hipMalloc3DArray(&d_proj1, &channelDesc, extent);
// hipMemcpy3DParms copyParams = {0};
// copyParams.extent = extent;
// copyParams.kind = hipMemcpyHostToDevice;
// copyParams.srcPtr = p_proj;
// copyParams.dstArray = d_proj;
// hipMemcpy3D(&copyParams);
// copyParams.srcPtr = p_proj1;
// copyParams.dstArray = d_proj1;
// hipMemcpy3D(&copyParams);

// const dim3 gridSize((na + BLOCKWIDTH - 1) / BLOCKWIDTH, (nb + BLOCKHEIGHT - 1) / BLOCKHEIGHT, (1 + BLOCKDEPTH - 1) / BLOCKDEPTH);
// const dim3 blockSize(BLOCKWIDTH,BLOCKHEIGHT, BLOCKDEPTH);
// kernel_add<<<gridSize, blockSize>>>(d_proj1, d_proj, iv, na, nb, -1);
// hipMemcpy(h_proj1, d_proj1, na * nb * sizeof(float), hipMemcpyDeviceToHost);
// hipFreeArray(d_proj);
// hipFreeArray(d_proj1);
// //Reset device for profiling
// hipDeviceReset();
// return;
// }