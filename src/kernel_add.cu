#include "hip/hip_runtime.h"
#include "kernel_add.cuh"

__global__ void kernel_add(float *proj1, float *proj, int iv, int na, int nb, float weight){
    int ia = BLOCKWIDTH * blockIdx.x + threadIdx.x;
    int ib = BLOCKHEIGHT * blockIdx.y + threadIdx.y;
    if (ia >= na || ib >= nb)
        return;
    proj1[ia + ib * na] += proj[ia + ib * na + iv * na * nb] * weight;
}

__global__ void kernel_add(float *proj1, float *proj, int na, int nb, float weight){
    int ia = BLOCKWIDTH * blockIdx.x + threadIdx.x;
    int ib = BLOCKHEIGHT * blockIdx.y + threadIdx.y;
    if (ia >= na || ib >= nb)
        return;
    proj1[ia + ib * na] += proj[ia + ib * na] * weight;
}

