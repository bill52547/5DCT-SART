#include "hip/hip_runtime.h"
__global__ void kernel_deformation(hipArray *singleViewImg1, hipTextureObject_t tex_img, hipArray *mx2, hipArray *my2, hipArray *mz2, int nx, int ny, int nz){
    int x = blockSize.x * blockIdx.x + threadIdx.x;
    int y = blockSize.y * blockIdx.y + threadIdx.y;
    int z = blockSize.z * blockIdx.z + threadIdx.z;
    if (x >= nx || y >= ny || z >= nz)
        return;
    int xi = mx2[x][y][z];
    int yi = my2[x][y][z];
    int zi = mz2[x][y][z];

    singleViewImg1[x][y][z] = tex3D<float>(tex_img, xi-0.5f, yi-0.5f, zi-0.5f);
}
