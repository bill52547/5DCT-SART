#include "hip/hip_runtime.h"
#include "kernel_update.cuh"
__global__ void kernel_update(float *img1, float *img, int nx, int ny, int nz, float lambda){
    int ix = BLOCKWIDTH * blockIdx.x + threadIdx.x;
    int iy = BLOCKHEIGHT * blockIdx.y + threadIdx.y;
    int iz = BLOCKDEPTH * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;
    int id = ix + iy * nx + iz * nx * ny;
    img1[id] -= lambda * img[id];
    if (img1[id] < 0.0f)
        img1[id] = 0.0f;
    if (img1[id] > 2500.0f)
        img1[id] = 0.0f;
}
