#include "hip/hip_runtime.h"
#include "kernel_backprojection.cuh"

__host__ void kernel_backprojection(float *d_img, float *d_proj, float angle,float SO, float SD, float da, int na, float ai, float db, int nb, float bi, int nx, int ny, int nz)
{
    hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
    struct hipExtent extent = make_hipExtent(na, nb, 1);
    hipArray *array_proj;
    hipMalloc3DArray(&array_proj, &channelDesc, extent);
    hipMemcpy3DParms copyParams = {0};
    hipPitchedPtr dp_proj = make_hipPitchedPtr((void*) d_proj, na * sizeof(float), na, nb);
    copyParams.extent = extent;
    copyParams.kind = hipMemcpyDeviceToDevice;
    copyParams.srcPtr = dp_proj;
    copyParams.dstArray = array_proj;
    hipMemcpy3D(&copyParams);

    hipResourceDesc resDesc;
    hipTextureDesc texDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;

    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0] = hipAddressModeBorder;
    texDesc.addressMode[1] = hipAddressModeBorder;
    texDesc.addressMode[2] = hipAddressModeBorder;
    texDesc.filterMode = hipFilterModeLinear;
    texDesc.readMode = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    resDesc.res.array.array = array_proj;
	hipTextureObject_t tex_proj = 0;
    hipCreateTextureObject(&tex_proj, &resDesc, &texDesc, NULL);

    const dim3 gridSize_img((nx + BLOCKWIDTH - 1) / BLOCKWIDTH, (ny + BLOCKHEIGHT - 1) / BLOCKHEIGHT, (nz + BLOCKDEPTH - 1) / BLOCKDEPTH);
    const dim3 blockSize(BLOCKWIDTH, BLOCKHEIGHT, BLOCKDEPTH);

    kernel<<<gridSize_img, blockSize>>>(d_img, tex_proj, angle, SO, SD, na, nb, da, db, ai, bi, nx, ny, nz);
    hipDeviceSynchronize();

    hipFreeArray(array_proj);
    hipDestroyTextureObject(tex_proj);
}

__global__ void kernel(float *img, hipTextureObject_t tex_proj, float angle, float SO, float SD, int na, int nb, float da, float db, float ai, float bi, int nx, int ny, int nz){
    int ix = BLOCKWIDTH * blockIdx.x + threadIdx.x;
    int iy = BLOCKHEIGHT * blockIdx.y + threadIdx.y;
    int iz = BLOCKDEPTH * blockIdx.z + threadIdx.z;
    if (ix >= nx || iy >= ny || iz >= nz)
        return;

    int id = ix + iy * nx + iz * nx * ny;

    img[id] = 0.0f;

	float sphi = __sinf(angle);
	float cphi = __cosf(angle);
	// float dd_voxel[3];
	float xc, yc, zc, xc0, yc0;
	xc0 = (float)ix - nx / 2 + 0.5f;
	yc0 = (float)iy - ny / 2 + 0.5f;
	zc = (float)iz - nz / 2 + 0.5f;
	xc = xc0 * cphi + yc0 * sphi;
	yc = -xc0 * sphi + yc0 * cphi;

	float x1, y1, z1;
	x1 = -SO;
	y1 = 0;
	z1 = 0;
	
	float x2, y2, z2;
	x2 = SD - SO;
	y2 = (x2 - x1) / (xc - x1) * (yc - y1) + y1;
	z2 = (x2 - x1) / (xc - x1) * (zc - z1) + z1;

	float a, b;
	a = y2 - ai + 0.5f;
	b = z2 - bi + 0.5f;
	
	img[id] += tex3D<float>(tex_proj, a, b, 0.5f);
}