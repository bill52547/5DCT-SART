#include "SART_cuda.h" // consists all required package and functions

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
// Macro for input and output
#define IN_IMG prhs[0]
#define PROJ prhs[1]
#define GEO_PARA prhs[2]
#define ITER_PARA prhs[3]
#define OUT_IMG plhs[0]

// load parameters
// assume all the parameter are orginized as:
// dx = dy = dz = 1 
// da = db

// load geometry parameters, all need parameter for single view projection
int nx, ny, nz, na, nb, numImg, numBytesImg, numSingleProj, numBytesSingleProj;
float da, db, ai, bi, SO, SD, dx;

// resolutions of volumes 
if (mxGetField(GEO_PARA, 0, "nx") != NULL)
    nx = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nx"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution nx.\n");

if (mxGetField(GEO_PARA, 0, "ny") != NULL)
    ny = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "ny"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution ny.\n");

if (mxGetField(GEO_PARA, 0, "nz") != NULL)
    nz = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nz"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid volume resolution nz.\n");

numImg = nx * ny * nz; // size of image
numBytesImg = numImg * sizeof(float); // number of bytes in image

// detector plane resolutions
if (mxGetField(GEO_PARA, 0, "na") != NULL)
    na = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "na"));
else if (mxGetField(GEO_PARA, 0, "nu") != NULL)
    na = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nu"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid number of detector in plane, which is denoted as na or nu.\n");

if (mxGetField(GEO_PARA, 0, "nb") != NULL)
    nb = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nb"));
else if (mxGetField(GEO_PARA, 0, "nv") != NULL)
    nb = (int)mxGetScalar(mxGetField(GEO_PARA, 0, "nv"));
else
	mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid number of detector across plane, which is denoted as nb or nv.\n");

numSingleProj = na * nb;
numBytesSingleProj = numSingleProj * sizeof(float);
if (mxGetField(GEO_PARA, 0, "dx") != NULL)
    dx = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "dx"));
else{
    dx = 1;
    mexPrintf("Automatically set voxel size dx to 1. \n");
    mexPrintf("If don't want that default value, please set para.dx manually.\n");
}
dx = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "dx"));
da = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "da")) / dx;
db = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "db")) / dx;

// detector plane offset from centered calibrations
if (mxGetField(GEO_PARA, 0, "ai") != NULL)
    ai = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "ai")) - (float)na / 2 + 0.5f;
else{
    mexPrintf("Automatically set detector offset ai to 0. \n");
    mexPrintf("If don't want that default value, please set para.ai manually.\n");
    ai = - (float)na / 2 + 0.5f;
}

if (mxGetField(GEO_PARA, 0, "bi") != NULL)
    bi = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "bi")) - (float)nb / 2 + 0.5f;
else{
    mexPrintf("Automatically set detector offset bi to 0. \n");
    mexPrintf("If don't want that default value, please set para.bi manually.\n");
    bi = - (float)nb / 2 + 0.5f;
}

if (mxGetField(GEO_PARA, 0, "SO") != NULL)
    SO = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SO")) / dx;
else if (mxGetField(GEO_PARA, 0, "DI") != NULL)
    SO = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "DI")) / dx;
else
    mexErrMsgIdAndTxt("MATLAB:badInput","Can't found valid distance between source and isocenter, which is denoted with para.SO or para.DI.\n");

SD = (float)mxGetScalar(mxGetField(GEO_PARA, 0, "SD")) / dx;

// load iterating parameters, for the whole bin
int n_view, n_iter, numProj, numBytesProj;
float *h_mx, *h_my, *h_mz, *h_mx2, *h_my2, *h_mz2, *angles, lambda;
n_view = (int)mxGetScalar(mxGetField(ITER_PARA, 0, "nv")); // number of views in this bin
n_iter = (int)mxGetScalar(mxGetField(ITER_PARA, 0, "n_iter")); // number of iterations of SART
h_mx = (float*)mxGetData(mxGetField(ITER_PARA, 0, "mx")); // index stead of difference
h_my = (float*)mxGetData(mxGetField(ITER_PARA, 0, "my")); 
h_mz = (float*)mxGetData(mxGetField(ITER_PARA, 0, "mz"));
h_mx2 = (float*)mxGetData(mxGetField(ITER_PARA, 0, "mx2")); // index stead of difference
h_my2 = (float*)mxGetData(mxGetField(ITER_PARA, 0, "my2")); 
h_mz2 = (float*)mxGetData(mxGetField(ITER_PARA, 0, "mz2"));
numProj = numSingleProj * n_view;
numBytesProj = numProj * sizeof(float);
angles = (float*)mxGetData(mxGetField(ITER_PARA, 0, "angles"));
lambda = (float)mxGetScalar(mxGetField(ITER_PARA, 0, "lambda"));
// load initial guess of image
float *h_img;
h_img = (float*)mxGetData(IN_IMG);

// load true projection value
float *h_proj;
h_proj = (float*)mxGetData(PROJ);

// define thread distributions
const dim3 gridSize_img((nx + BLOCKWIDTH - 1) / BLOCKWIDTH, (ny + BLOCKHEIGHT - 1) / BLOCKHEIGHT, (nz + BLOCKDEPTH - 1) / BLOCKDEPTH);
const dim3 gridSize_singleProj((na + BLOCKWIDTH - 1) / BLOCKWIDTH, (nb + BLOCKHEIGHT - 1) / BLOCKHEIGHT, 1);
const dim3 gridSize_proj((na + BLOCKWIDTH - 1) / BLOCKWIDTH, (nb + BLOCKHEIGHT - 1) / BLOCKHEIGHT, (n_view + BLOCKDEPTH - 1) / BLOCKDEPTH);
const dim3 blockSize(BLOCKWIDTH,BLOCKHEIGHT, BLOCKDEPTH);

// CUDA 3DArray Malloc parameters
hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
struct hipExtent extent_img = make_hipExtent(nx, ny, nz);
struct hipExtent extent_proj = make_hipExtent(na, nb, n_view);
struct hipExtent extent_singleProj = make_hipExtent(na, nb, 1);

// malloc in device: DVF for SINGLE view image from the bin
float *d_mx, *d_my, *d_mz, *d_mx2, *d_my2, *d_mz2;
hipMalloc((void**)&d_mx, numBytesImg);
hipMalloc((void**)&d_my, numBytesImg);
hipMalloc((void**)&d_mz, numBytesImg);
hipMalloc((void**)&d_mx2, numBytesImg);
hipMalloc((void**)&d_my2, numBytesImg);
hipMalloc((void**)&d_mz2, numBytesImg);

// malloc in device: projection of the whole bin
float *d_proj;
hipMalloc((void**)&d_proj, numBytesProj);

// copy to device: projection of the whole bin
hipMemcpy(d_proj, h_proj, numBytesProj, hipMemcpyHostToDevice);

// malloc in device: another projection pointer, with single view size
float *d_singleViewProj2;
hipMalloc((void**)&d_singleViewProj2, numBytesSingleProj);

// malloc in device: projection of the whole bin
float *d_img;
hipMalloc((void**)&d_img, numBytesImg);

// copy to device: initial guess of image
hipMemcpy(d_img, h_img, numBytesImg, hipMemcpyHostToDevice);

// malloc in device: another image pointer, for single view 
float *d_singleViewImg1, *d_singleViewImg2, *d_imgOnes;
hipMalloc(&d_singleViewImg1, numBytesImg);
hipMalloc(&d_singleViewImg2, numBytesImg);
hipMalloc(&d_imgOnes, numBytesImg);
float angle;
for (int iter = 0; iter < n_iter; iter++){ // iteration
    for (int i_view = 0; i_view < n_view; i_view++){ // view
        // mexPrintf("i_view = %d.\n", i_view);
        
        angle = angles[i_view];
        // memory copy to device of: DVF from bin reference image to i_view image
        // X
        hipMemcpy(d_mx, h_mx + i_view * numImg, numBytesImg, hipMemcpyHostToDevice);

        // Y
        hipMemcpy(d_my, h_my + i_view * numImg, numBytesImg, hipMemcpyHostToDevice);

        // Z
        hipMemcpy(d_mz, h_mz + i_view * numImg, numBytesImg, hipMemcpyHostToDevice);

        // memory copy to device of: inverted DVF from bin reference image to i_view image
        // X
        hipMemcpy(d_mx2, h_mx2 + i_view * numImg, numBytesImg, hipMemcpyHostToDevice);

        // Y
        hipMemcpy(d_my2, h_my2 + i_view * numImg, numBytesImg, hipMemcpyHostToDevice);

        // Z
        hipMemcpy(d_mz2, h_mz2 + i_view * numImg, numBytesImg, hipMemcpyHostToDevice);
        

        // deformed image for i_view, from reference image of the bin
        kernel_deformation<<<gridSize_img, blockSize>>>(d_singleViewImg1, d_img, d_mx2, d_my2, d_mz2, nx, ny, nz);
        hipDeviceSynchronize();

        // projection of deformed image from initial guess
        kernel_projection<<<gridSize_singleProj, blockSize>>>(d_singleViewProj2, d_singleViewImg1, angle, SO, SD, da, na, ai, db, nb, bi, nx, ny, nz); // TBD
        hipDeviceSynchronize();

        // difference between true projection and projection from initial guess
        // update d_singleViewProj2 instead of malloc a new one
        kernel_add<<<gridSize_singleProj, blockSize>>>(d_singleViewProj2, d_proj, i_view, na, nb, -1);
        hipDeviceSynchronize();

        // backprojecting the difference of projections
        kernel_backprojection<<<gridSize_img, blockSize>>>(d_singleViewImg1, d_singleViewProj2, angle, SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
        hipDeviceSynchronize();

        // deform backprojection back to the bin
        kernel_deformation<<<gridSize_img, blockSize>>>(d_singleViewImg2, d_singleViewImg1, d_mx, d_my, d_mz, nx, ny, nz);
        hipDeviceSynchronize();

        // calculate the ones backprojection data
        kernel_initial<<<gridSize_img, blockSize>>>(d_singleViewImg1, nx, ny, nz, 1);
        hipDeviceSynchronize();

        kernel_projection<<<gridSize_singleProj, blockSize>>>(d_singleViewProj2, d_singleViewImg1, angle, SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
        hipDeviceSynchronize();

        kernel_backprojection<<<gridSize_img, blockSize>>>(d_singleViewImg1, d_singleViewProj2, angle, SO, SD, da, na, ai, db, nb, bi, nx, ny, nz);
        hipDeviceSynchronize();

        // weighting
        kernel_division<<<gridSize_img, blockSize>>>(d_singleViewImg2, d_singleViewImg1, nx, ny, nz);
        hipDeviceSynchronize();
        
        // updating
        kernel_update<<<gridSize_img, blockSize>>>(d_img, d_singleViewImg2, nx, ny, nz, lambda);
        hipDeviceSynchronize();              
    }
}
OUT_IMG = mxCreateNumericMatrix(0, 0, mxSINGLE_CLASS, mxREAL);



// const mwSize *outDim = mxGetDimensions(PROJ); // IN_IMG or PROJ
// mxSetDimensions(OUT_IMG, outDim, 3);
// mxSetData(OUT_IMG, mxMalloc(numBytesImg));
// float *h_outimg = (float*)mxGetData(OUT_IMG);
// hipMemcpy(h_outimg, d_singleViewProj2, numBytesSingleProj, hipMemcpyDeviceToHost);

const mwSize *outDim = mxGetDimensions(IN_IMG); // IN_IMG or PROJ
mxSetDimensions(OUT_IMG, outDim, 3);
mxSetData(OUT_IMG, mxMalloc(numBytesImg));
float *h_outimg = (float*)mxGetData(OUT_IMG);
hipMemcpy(h_outimg, d_img, numBytesImg, hipMemcpyDeviceToHost);

hipFree(d_mx);
hipFree(d_my);
hipFree(d_mz);
hipFree(d_mx2);
hipFree(d_my2);
hipFree(d_mz2);
// hipFreeArray(d_img);
hipFree(d_proj);
hipFree(d_singleViewImg1);
hipFree(d_singleViewImg2);
hipFree(d_singleViewProj2);

hipFree(d_img);
hipDeviceReset();
return;
}

